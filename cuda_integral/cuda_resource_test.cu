#include <iostream>
#include <option_parser/ConfigFileOpt.h>

#define COEF_NUM 5
#define gpuErrorCheck(ans); { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

__constant__ double c[COEF_NUM], a1[COEF_NUM], a2[COEF_NUM];

int main(int argc, char *argv[]) {
//  //////////////////////////// Program Parameter Parsing ////////////////////////////
    std::string file_name = "execution.conf";

//  ////////////////////////////    Config File Parsing    ////////////////////////////
    ConfigFileOpt config{};
    try {
        config.parse(file_name);
    } catch (std::exception &ex) {
        std::cerr << "Error: " << ex.what() << std::endl;
        return 3;
    }

    //  ////////////////////////////   Integration Initiation   ////////////////////////////

    double *d_a1;gpuErrorCheck(hipMalloc(&d_a1, sizeof(double) * COEF_NUM))

    // Copy host vectors to device
    gpuErrorCheck(hipMemcpy(d_a1, &config.get_a1()[0], sizeof(double) * COEF_NUM, hipMemcpyHostToDevice))
//    gpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(&a1), &config.get_a1()[0], sizeof(double) * COEF_NUM, hipMemcpyHostToDevice));

    double a1_out[COEF_NUM]; //
    gpuErrorCheck(hipMemcpy(a1_out, d_a1, sizeof(double) * COEF_NUM, hipMemcpyDeviceToHost));


    for (int i = 0; i < COEF_NUM; ++i)
        std::cout << a1_out[i] << "  ==  " << config.get_a1()[i] << std::endl;

    return 0;
}
